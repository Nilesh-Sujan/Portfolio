/* INSTRUCTIONS FOR CHANGING INPUTSIZE
 * UNDER GLOBAL DEFINITIONS, CHANGE "inputSize" VARIABLE.
 */

// Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Global definitions for ease of access
int blockThreads_num = 1024; // Initialize variable for number of threads per block
int inputSize = 8;           // Initialize variable for list input size. 8/1024/1025/65535

// Function that runs on GPU to reduce the array
__global__ void reduce(int *input, int *output, int deviceinputSize)
{
    extern __shared__ int sdata[]; // Dynamically allocated shared memory

    unsigned int tid = threadIdx.x;                               // Define threadID
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x; // Index in the grid

    if (i < deviceinputSize)
    { // Check if there is more elements in the dataset than threads
        // perform first level of reduction,
        // reading from global memory, writing to shared memory
        // OPTIMIZATION: Deals with idle threads
        sdata[tid] = input[i] + input[i + blockDim.x]; // Each thread loads one element from global to shared memory

        __syncthreads(); // Synchronizes to protect from read-after-write memory race conditions within the block. Waits for code to reach this point.

        // Performs the reduction in shared memory.
        // OPTIMIZATION: reversed loop and threadID-based indexing to use sequential addressing rather than interleaved addressing
        for (unsigned int s = 1; s < blockDim.x; s *= 2)
        {
            if (tid % (2 * s) == 0)
            {
                sdata[tid] += sdata[tid + s];
            }

            __syncthreads(); // Synchronizes to protect from read-after-write memory race conditions within the block. Waits for code to reach this point.
        }

        // Writes result for this block to global memory
        if (tid == 0)
            output[blockIdx.x] = sdata[0];
    }
}

// Helper function that runs on CPU to fills in the array with all the numbers up to inputarraylength.
void fill_array(int *a, int n)
{
    for (int i = 0; i < n; i++) // For loop through the array size
        a[i] = i;               // Fill in each array index with the index of the loop
}

// Helper function that calculates the appropriate output size for the output array based on input size
int initOutputArray(int inputSize, int outputSize)
{
    outputSize = inputSize / (blockThreads_num / 2); // Divide input size by the amount of thread block going to be used
    if (inputSize % (blockThreads_num / 2))
    {                 // Check if the inputsize is divisible by the specificed number of threads in a block.
        outputSize++; // Increment the output size
    }
    return outputSize; // Return output size
}

// Helper function that calculates the sum of the reduced array
int calculateSum(int *hostOutput, int outputSize)
{
    for (int i = 1; i < outputSize; i++)
    {                                   // Loop through the output array
        hostOutput[0] += hostOutput[i]; // Perform the calculation while appending it to the first element.
    }

    return hostOutput[0]; // Return the first element
}

// Main function, entry point of program.
int main()
{
    // Definitions
    int *a;             // Define input host array
    int *a_out;         // Define output host array
    int *b;             // Define input device array
    int *b_out;         // Define output device array
    int outputSize = 0; // Define output size of output array

    outputSize = initOutputArray(inputSize, outputSize); // Inititalize output array

    a = (int *)malloc(inputSize * sizeof(int));      // Allocate memory to host input array
    a_out = (int *)malloc(outputSize * sizeof(int)); // Allocate memory to host output array

    if (a == NULL || a_out == NULL)
    {
        printf("Failed to allocate memory to host array!\n");
        exit(EXIT_FAILURE);
    }

    fill_array(a, inputSize); // Fill in the array with the appropriate numbers

    dim3 blockSize(blockThreads_num, 1, 1); // Create blocksize based on threadnum
    dim3 gridSize(outputSize, 1, 1);        // Create gridsize based on outputSize

    hipError_t error = hipSuccess; // Error code to check return values for CUDA calls

    error = hipMalloc((void **)&b, inputSize * sizeof(int)); // Allocate memory to device input array

    if (error != hipSuccess) // Error checking
    {
        fprintf(stderr, "Failed to allocate memory to input device array. (Error Code: %s)\n", hipGetErrorString(error)); // Print error message if error
        exit(EXIT_FAILURE);                                                                                                // Exit program
    }

    error = hipMalloc((void **)&b_out, outputSize * sizeof(int)); // Allocate memory to device output array

    if (error != hipSuccess) // Error checking
    {
        fprintf(stderr, "Failed to allocate memory to output device array. (Error Code: %s)\n", hipGetErrorString(error)); // Print error message if error
        exit(EXIT_FAILURE);                                                                                                 // Exit program
    }

    error = hipMemcpy(b, a, inputSize * sizeof(int), hipMemcpyHostToDevice); // Copy data between the host and the device.

    if (error != hipSuccess) // Error checking
    {
        fprintf(stderr, "Failed to copy data from host to device (Error Code: %s)\n", hipGetErrorString(error)); // Print error message if error
        exit(EXIT_FAILURE);                                                                                       // Exit program
    }

    reduce<<<gridSize, blockSize, blockThreads_num * sizeof(double)>>>(b, b_out, inputSize); // Kernel for the reduce function containing gridsize, block size and amount of dynamically allocated shared memory

    error = hipGetLastError(); // Returns the last error from a runtime call.

    if (error != hipSuccess) // Error checking
    {
        fprintf(stderr, "Failed to launch kernel (Error Code: %s)\n", hipGetErrorString(error)); // Print error message if error
        exit(EXIT_FAILURE);                                                                       // Exit program
    }

    error = hipMemcpy(a_out, b_out, outputSize * sizeof(int), hipMemcpyDeviceToHost); // Copy data between the device and the host.

    if (error != hipSuccess) // Error checking
    {
        fprintf(stderr, "Failed to copy data from device to host (Error Code: %s)\n", hipGetErrorString(error)); // Print error message if error
        exit(EXIT_FAILURE);                                                                                       // Exit program
    }

    int sum = calculateSum(a_out, outputSize); // Accumulate the sum from the host output

    printf("Sum of list: %d\n", sum); // Print out the final sum

    error = hipFree(b); // Free device input memory

    if (error != hipSuccess) // Error checking
    {
        fprintf(stderr, "Failed to free device input memory (error code %s)!\n", hipGetErrorString(error)); // Print error message if error
        exit(EXIT_FAILURE);                                                                                  // Exit program
    }

    error = hipFree(b_out); // Free device output memory

    if (error != hipSuccess) // Error checking
    {
        fprintf(stderr, "Failed to free device output memory (error code %s)!\n", hipGetErrorString(error)); // Print error message if error
        exit(EXIT_FAILURE);                                                                                   // Exit program
    }

    free(a);     // Free host input array memory
    free(a_out); // Free host output array memory

    error = hipDeviceReset(); // Reset the device and exit

    if (error != hipSuccess) // Error checking
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(error)); // Print error message if error
        exit(EXIT_FAILURE);                                                                          // Exit program
    }

    hipDeviceSynchronize(); // Ensures that the GPU finishes before exiting as kernel execution is asynchronous.
}